
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}

/* experiment with N */
/* how large can it be? */
#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main()
{
    int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	/* allocate space for device copies of a, b, c */

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* allocate space for host copies of a, b, c and setup input values */

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	/* copy inputs to device */
	/* fix the parameters needed to copy data to the device */
	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */ 
	add<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );

	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );


	printf( "c[0] = %d\n", c[0] );
	printf( "c[1] = %d\n", c[1] );
	printf( "c[2] = %d\n", c[2] );
	printf( "c[3] = %d\n", c[3] );
	printf( "c[4] = %d\n", c[4] );

	printf( "c[%d] = %d\n", c[N-1] );

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
