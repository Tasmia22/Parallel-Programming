#include <stdio.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 512
#define MAX 50000
#define COEFFICIENT 1 // Example coefficient value

// Function to initialize coefficients array
void initialize(int coeffArr[], int size) {
    for (int i = 0; i < size; i++) {
        coeffArr[i] = COEFFICIENT;
    }
}

// Device function to compute power iteratively
__device__ double power(double base, int exp) {
    double result = 1.0;
    for (int i = 0; i < exp; i++) {
        result *= base;
    }
    return result;
}

// Kernel function to evaluate polynomial terms
__global__ void evaluatePolynomial(double x, int *coeffArr, double *outputTerms, int numCoefficients) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numCoefficients) {
        double termValue = power(x, idx); // Compute x^degree
        outputTerms[idx] = coeffArr[idx] * termValue; // Multiply by coefficient
    }
}

int main() {
    double x =.9999; // Example x value
    int *coeffArr = new int[MAX];
    int numCoefficients = MAX;

    initialize(coeffArr, numCoefficients);

    // Host memory for output terms
    double *outputTerms = (double *)malloc(numCoefficients * sizeof(double));

    // Device memory pointers
    int *d_coeffArr;
    double *d_outputTerms;

    // Allocate device memory
    hipMalloc((void **)&d_coeffArr, numCoefficients * sizeof(int));
    hipMalloc((void **)&d_outputTerms, numCoefficients * sizeof(double));

    // Copy coefficients to device
    hipMemcpy(d_coeffArr, coeffArr, numCoefficients * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int blocks = (numCoefficients + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    evaluatePolynomial<<<blocks, THREADS_PER_BLOCK>>>(x, d_coeffArr, d_outputTerms, numCoefficients);

    // Copy results back to host
    hipMemcpy(outputTerms, d_outputTerms, numCoefficients * sizeof(double), hipMemcpyDeviceToHost);

    // Compute final polynomial value on host
    double finalResult = 0;
    for (int i = 0; i < numCoefficients; i++) {
        finalResult += outputTerms[i];
    }

    // Print results
    printf("Final polynomial value: %.2f\n", finalResult);

    // Free memory
    free(outputTerms);
    hipFree(d_coeffArr);
    hipFree(d_outputTerms);
    delete[] coeffArr;

    return 0;
}
